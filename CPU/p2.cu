
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define size 224
#define block_size 14
#define P 16

__global__ void matrix_mult(int *a, int *b, int *c)
{
    int row = threadIdx.x;
    int col = threadIdx.y;

    int my_x = blockIdx.x * blockDim.x + threadIdx.x;
    int my_y = blockIdx.y * blockDim.y + threadIdx.y;

    int i, j;
    int local_c = 0;

    __shared__ int a_shared[P][P];
    __shared__ int b_shared[P][P];

    for (i = 0; i < size / block_size; i++)
    {
        a_shared[row][col] = a[my_x * size + (i * block_size + col)];
        b_shared[row][col] = b[(i * block_size + row) * size + my_y];
        __syncthreads();
        for (j = 0; j < block_size; j++)
        {
            local_c += a_shared[row][j] * b_shared[j][col];
        }
        __syncthreads();
    }

    c[my_x * size + my_y] = local_c;
}

int main()
{
    int i;
    int *a = (int *)malloc(sizeof(int) * size * size);
    int *b = (int *)malloc(sizeof(int) * size * size);
    int *c = (int *)malloc(sizeof(int) * size * size);

    for (i = 0; i < size * size; i++)
    {
        a[i] = 1;
        b[i] = 2;
        c[i] = 0;
    }

    int *gpu_a, *gpu_b, *gpu_c;
    hipMalloc((void **)&gpu_a, sizeof(int) * size * size);
    hipMalloc((void **)&gpu_b, sizeof(int) * size * size);
    hipMalloc((void **)&gpu_c, sizeof(int) * size * size);

    struct timespec start, stop;
    double time;

    hipMemcpy(gpu_a, a, sizeof(int) * size * size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, sizeof(int) * size * size, hipMemcpyHostToDevice);

    dim3 dimGrid(1, 1);  // 2 dimensional grid
    dim3 dimBlock(P, P); // 2 dimensional blocks

    if (clock_gettime(CLOCK_REALTIME, &start) == -1)
    {
        perror("clock gettime");
    }
    matrix_mult<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);
    hipMemcpy(c, gpu_c, sizeof(int) * size * size, hipMemcpyDeviceToHost);

    if (clock_gettime(CLOCK_REALTIME, &stop) == -1)
    {
        perror("clock gettime");
    }
    time = (stop.tv_sec - start.tv_sec) + (double)(stop.tv_nsec - start.tv_nsec) / 1e9;
    printf("time is %f ns\n", time * 1e9);

    printf("c[%d][%d]=%d ", 451, 451, c[451 * size + 451]);

    free(a);
    free(b);
    free(c);
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);
    return 0;
}